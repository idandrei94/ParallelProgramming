/* A small program initializing arrays with different values.
 *
 * This version uses 1D grids and 1D blocks.
 *
 * kernelConstant:	  using a constant
 * kernelBlockIdx:	  using blockIdx.x
 * kernelThreadIdx:	  using threadIdx.x
 * kernelGlobalThreadIdx: using the global thread index
 *
 *
 * Compiling:
 *   nvcc -arch=sm_50 -o index_1 index_1.cu
 *   clang --cuda-gpu-arch=sm_50 -o index_1 index_1.cu -lcudart
 *
 * Running:
 *   ./index_1
 *
 *
 * File: index_1.cu			Author: S. Gross
 * Date: 14.02.2018
 *
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCKS_PER_GRID   2
#define THREADS_PER_BLOCK 4
#define VECTOR_SIZE	  BLOCKS_PER_GRID * THREADS_PER_BLOCK


__global__ void kernelConstant (int *a, const size_t vecSize);
__global__ void kernelBlockIdx (int *a, const size_t vecSize);
__global__ void kernelThreadIdx (int *a, const size_t vecSize);
__global__ void kernelGlobalThreadIdx (int *a, const size_t vecSize);



/* define macro to check the return value of a CUDA function		*/
#define CheckRetValueOfCudaFunction(val) \
  if (val != hipSuccess) \
  { \
    fprintf (stderr, "file: %s  line %d: %s.\n", \
	     __FILE__, __LINE__, hipGetErrorString (val)); \
    hipDeviceReset (); \
    exit (EXIT_FAILURE); \
  }
    

int main (void)
{
  int aConstant[VECTOR_SIZE],		/* arrays on CPU		*/
      aBlockIdx[VECTOR_SIZE],
      aThreadIdx[VECTOR_SIZE],
      aGlobalThreadIdx[VECTOR_SIZE],
      *dev_aConstant,			/* array addresses on device	*/
      *dev_aBlockIdx,
      *dev_aThreadIdx,
      *dev_aGlobalThreadIdx;
  hipError_t ret;			/* CUDA function return value	*/


  /* allocate memory for all arrays on the GPU (device)			*/
  ret = hipMalloc ((void **) &dev_aConstant,
		    VECTOR_SIZE * sizeof (int));
  CheckRetValueOfCudaFunction (ret);
  ret = hipMalloc ((void **) &dev_aBlockIdx,
		    VECTOR_SIZE * sizeof (int));
  CheckRetValueOfCudaFunction (ret);
  ret = hipMalloc ((void **) &dev_aThreadIdx,
		    VECTOR_SIZE * sizeof (int));
  CheckRetValueOfCudaFunction (ret);
  ret = hipMalloc ((void **) &dev_aGlobalThreadIdx,
		    VECTOR_SIZE * sizeof (int));
  CheckRetValueOfCudaFunction (ret);

  /* run all kernels concurrently					*/
  kernelConstant <<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>
    (dev_aConstant, VECTOR_SIZE);
  kernelBlockIdx <<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>
    (dev_aBlockIdx, VECTOR_SIZE);
  kernelThreadIdx <<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>
    (dev_aThreadIdx, VECTOR_SIZE);
  kernelGlobalThreadIdx <<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>
    (dev_aGlobalThreadIdx, VECTOR_SIZE);

  /* copy initialized arrays back from the GPU to the CPU		*/
  ret = hipMemcpy (aConstant, dev_aConstant,
		    VECTOR_SIZE * sizeof (int), hipMemcpyDeviceToHost);
  CheckRetValueOfCudaFunction (ret);
  ret = hipMemcpy (aBlockIdx, dev_aBlockIdx,
		    VECTOR_SIZE * sizeof (int), hipMemcpyDeviceToHost);
  CheckRetValueOfCudaFunction (ret);
  ret = hipMemcpy (aThreadIdx, dev_aThreadIdx,
		    VECTOR_SIZE * sizeof (int), hipMemcpyDeviceToHost);
  CheckRetValueOfCudaFunction (ret);
  ret = hipMemcpy (aGlobalThreadIdx, dev_aGlobalThreadIdx,
		    VECTOR_SIZE * sizeof (int), hipMemcpyDeviceToHost);
  CheckRetValueOfCudaFunction (ret);

  /* print results							*/
  printf ("Initialization with a constant\n");
  for (int i = 0; i < VECTOR_SIZE; ++i)
  {
    printf ("  %d", aConstant[i]);
  }
  printf ("\n\nInitialization with the blockIdx\n");
  for (int i = 0; i < VECTOR_SIZE; ++i)
  {
    printf ("  %d", aBlockIdx[i]);
  }
  printf ("\n\nInitialization with the threadIdx\n");
  for (int i = 0; i < VECTOR_SIZE; ++i)
  {
    printf ("  %d", aThreadIdx[i]);
  }
  printf ("\n\nInitialization with the global thread index\n");
  for (int i = 0; i < VECTOR_SIZE; ++i)
  {
    printf ("  %d", aGlobalThreadIdx[i]);
  }
  printf ("\n\n");

  /* free allocated memory on the GPU					*/
  ret = hipFree (dev_aConstant);
  CheckRetValueOfCudaFunction (ret);
  ret = hipFree (dev_aBlockIdx);
  CheckRetValueOfCudaFunction (ret);
  ret = hipFree (dev_aThreadIdx);
  CheckRetValueOfCudaFunction (ret);
  ret = hipFree (dev_aGlobalThreadIdx);
  CheckRetValueOfCudaFunction (ret);

  /* reset current device						*/
  ret = hipDeviceReset ();
  CheckRetValueOfCudaFunction (ret);

  return EXIT_SUCCESS;
}


/* Initialize vector "a" with different values using GPU threads.
 *
 * Input:		vecSize		vector size
 * Output		a		initialized array
 * Return value:	none
 * Sideeffects:		none
 *
 */
__global__ void kernelConstant (int *a, const size_t vecSize)
{
  int idx = (int) (blockIdx.x * blockDim.x + threadIdx.x);

  if (idx < (int) vecSize)
  {
    a[idx] = 9;
  }
}


__global__ void kernelBlockIdx (int *a, const size_t vecSize)
{
  int idx = (int) (blockIdx.x * blockDim.x + threadIdx.x);

  if (idx < (int) vecSize)
  {
    a[idx] = (int) blockIdx.x;
  }
}


__global__ void kernelThreadIdx (int *a, const size_t vecSize)
{
  int idx = (int) (blockIdx.x * blockDim.x + threadIdx.x);

  if (idx < (int) vecSize)
  {
    a[idx] = (int) threadIdx.x;
  }
}


__global__ void kernelGlobalThreadIdx (int *a, const size_t vecSize)

{
  int idx = (int) (blockIdx.x * blockDim.x + threadIdx.x);

  if (idx < (int) vecSize)
  {
    a[idx] = idx;
  }
}
