/* This program shows branch divergence in warps, if you split the
 * threads in odd / even warps.
 *
 *
 * Compiling:
 *   nvcc -arch=sm_50 -o divergence_2 divergence_2.cu
 *   clang --cuda-gpu-arch=sm_50 -o divergence_2 divergence_2.cu -lcudart
 *
 * Running:
 *   ./divergence_2
 *
 *
 * File: divergence_2.cu		Author: S. Gross
 * Date: 14.02.2018
 *
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCKS_PER_GRID   1
#define THREADS_PER_BLOCK 64		/* two warps			*/


/* define macro to check the return value of a CUDA function		*/
#define CheckRetValueOfCudaFunction(val) \
  if (val != hipSuccess) \
  { \
    fprintf (stderr, "file: %s  line %d: %s.\n", \
	     __FILE__, __LINE__, hipGetErrorString (val)); \
    hipDeviceReset (); \
    exit (EXIT_FAILURE); \
  }

__global__ void showDivergence (void);


int main (void)
{
  hipError_t ret;			/* CUDA function return value   */

  showDivergence <<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>> ();

  /* reset current device						*/
  ret = hipDeviceReset ();
  CheckRetValueOfCudaFunction (ret);

  return EXIT_SUCCESS;
}


__global__ void showDivergence (void)
{
  int warpNumber = threadIdx.x / warpSize;

  /* warpNumber can also be used in a switch-statment, if different
   * warps should do different work.
   */
  if ((warpNumber & 1) == 0)
  {
    /* even thread numbers						*/
    printf ("Thread %d in warp %d from thread block %d doing "
	    "work 1.\n",
	  threadIdx.x, warpNumber, blockIdx.x);
  }
  else
  {
    /* even thread numbers						*/
    printf ("Thread %d in warp %d from thread block %d doing "
	    "work 2.\n",
	  threadIdx.x, warpNumber, blockIdx.x);
  }
}
